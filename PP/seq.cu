
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define IDX2C(i,j,ld) (((j)*(ld))+(i)) 
#define SIZE 5600 // 1400, 2800, 5600

void cpu_mmul(const float* A, const float* B, float* C, int m_lenght) {
    for (int i = 0; i < m_lenght; ++i) {
        for (int j = 0; j < m_lenght; ++j) {
            C[IDX2C(i, j, m_lenght)] = 0.0;
            for (int r = 0; r < m_lenght; ++r) {
                C[IDX2C(i, j, m_lenght)] += A[IDX2C(i, r, m_lenght)] * B[IDX2C(r, j, m_lenght)];
            }
        }
    }
}


void print_matrix(float* matrix, int m_lenght) {
    printf("\n");
    for (int i = 0; i < m_lenght; ++i) {
        for (int j = 0; j < m_lenght; ++j) {
            printf("%f ", matrix[j * m_lenght + i]);
        }
        printf("\n");
    }
    printf("\n");
}


int main() {
    int m_lenght = SIZE, n2b = m_lenght * m_lenght * sizeof(float);
    float* h_A = (float*)malloc(n2b);
    float* h_B = (float*)malloc(n2b);
    float* h_C = (float*)malloc(n2b);

    double start_time, end_time, timeofwork;
    srand(time(0));

    start_time = clock();
    for (int i = 0; i < 12; ++i) {
        for (int j = 0; j < m_lenght * m_lenght; j++) {
            h_A[j] = (float)rand() / RAND_MAX;
            h_B[j] = (float)rand() / RAND_MAX;
        }

        cpu_mmul(h_A, h_B, h_C, m_lenght);
    }
    end_time = clock();
    timeofwork = end_time - start_time;
    printf("Sequence. SIZE: %d\n", SIZE);
    printf("TIME OF WORK IS: %.9f\n\n", timeofwork / CLOCKS_PER_SEC / 12);

    printf("A =\n");
    print_matrix(h_A, 5);
    printf("B =\n");
    print_matrix(h_B, 5);

    printf("C =\n");
    print_matrix(h_C, 5);

    free(h_A);
    free(h_B);
    free(h_C);
    return 0;
}
