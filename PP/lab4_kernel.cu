#include "hip/hip_runtime.h"
#define kernel addKernel
#include "mainGPU.h"

__global__ 
void addKernel(double* c, double* a, double* b, unsigned int size) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x)
        c[i] = a[i] + b[i];
}