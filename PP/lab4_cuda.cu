#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <malloc.h>
#include <stdio.h>
#include <stdlib.h>

__global__ 
void addKernel(double* c, double* a, double* b, unsigned int size) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x)
        c[i] = a[i] + b[i];
}

int main(int argc, char* argv[]) {

    printf("\n GridSize = %d \n", GRID_SIZE);
    printf("BlockSize = %d \n", BLOCK_SIZE);
    printf("N = %d\n", N);

    int n2b = N * sizeof(double);
	int n2=N;
	
    // Выделение памяти на хосте
   	double * a=(double*)calloc(n2,sizeof(double));
	double * b=(double*)calloc(n2,sizeof(double));
	double * c=(double*)calloc(n2,sizeof(double));
	// Инициализация массивов
    for (int i = 0; i < N; ++i){
		a[i] = 1.1;
		b[i] = 1.5;
	}
    // Выделение памяти на устройстве
    double* adev = NULL;
    hipError_t cuerr = hipMalloc((void**)&adev, n2b);
    if (cuerr != hipSuccess)
    {
        fprintf(stderr, "Cannot allocate device array for a: %s\n",
            hipGetErrorString(cuerr));
        return 0;
    }

    double* bdev = NULL;
    cuerr = hipMalloc((void**)&bdev, n2b);
    if (cuerr != hipSuccess)
    {
        fprintf(stderr, "Cannot allocate device array for b: %s\n",
            hipGetErrorString(cuerr));
        return 0;
    }

    double * cdev = NULL;
    cuerr = hipMalloc((void**)&cdev, n2b);
    if (cuerr != hipSuccess)
    {
        fprintf(stderr, "Cannot allocate device array for c: %s\n",
            hipGetErrorString(cuerr));
        return 0;
    }

    // Создание обработчиков событий
    hipEvent_t start, stop;
    float gpuTime = 0.0f;
    cuerr = hipEventCreate(&start);
    if (cuerr != hipSuccess)
    {
        fprintf(stderr, "Cannot create CUDA start event: %s\n",
            hipGetErrorString(cuerr));
        return 0;
    }

    cuerr = hipEventCreate(&stop);
    if (cuerr != hipSuccess)
    {
        fprintf(stderr, "Cannot create CUDA end event: %s\n",
            hipGetErrorString(cuerr));
        return 0;
    }

	// Копирование данных с хоста на девайс
    cuerr = hipMemcpy(adev, a, n2b, hipMemcpyHostToDevice);
    if (cuerr != hipSuccess)
    {
        fprintf(stderr, "Cannot copy a array from host to device: %s\n",
            hipGetErrorString(cuerr));
        return 0;
    }

    cuerr = hipMemcpy(bdev, b, n2b, hipMemcpyHostToDevice);
    if (cuerr != hipSuccess)
    {
        fprintf(stderr, "Cannot copy b array from host to device: %s\n",
            hipGetErrorString(cuerr));
        return 0;
    }

    // Установка точки старта
    cuerr = hipEventRecord(start, 0);
    if (cuerr != hipSuccess)
    {
        fprintf(stderr, "Cannot record CUDA event: %s\n",
            hipGetErrorString(cuerr));
        return 0;
    }

    //Запуск ядра
    addKernel<<< GRID_SIZE, BLOCK_SIZE >>>(cdev, adev, bdev, N);

    cuerr = hipGetLastError();
    if (cuerr != hipSuccess)
    {
        fprintf(stderr, "Cannot launch CUDA kernel: %s\n",
            hipGetErrorString(cuerr));
        return 0;
    }

	// Синхронизация устройств
    cuerr = hipDeviceSynchronize();
    if (cuerr != hipSuccess)
    {
        fprintf(stderr, "Cannot synchronize CUDA kernel: %s\n",
            hipGetErrorString(cuerr));
        return 0;
    }

	// Установка точки окончания
    cuerr = hipEventRecord(stop, 0);
    if (cuerr != hipSuccess)
    {
        fprintf(stderr, "Cannot copy c array from device to host: %s\n",
            hipGetErrorString(cuerr));
        return 0;
    }

	// Копирование результата на хост
    cuerr = hipMemcpy(c, cdev, n2b, hipMemcpyDeviceToHost);
    if (cuerr != hipSuccess)
    {
        fprintf(stderr, "Cannot copy c array from device to host: %s\n",
            hipGetErrorString(cuerr));
        return 0;
    }

	// Расчет времени
    cuerr = hipEventElapsedTime(&gpuTime, start, stop);
    printf("time spent executing %s: %.9f seconds\n", "kernel", gpuTime/1000);

    // Очищение памяти
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(adev);
    hipFree(bdev);
    hipFree(cdev);
    free(a);
    free(b);
    free(c);
};